#include "hip/hip_runtime.h"
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iostream>
#include <cmath>
#include <sstream>
#include<Windows.h>

#define ANTS 1024
#define ALPHA 2
#define BETA 10
#define RHO 0.5
#define Q 50
#define MAX_ITERATIONS 10

#define NODES 105
#define DIST 10000
#define PHERO_INITIAL (1.0 / NODES)
#define TOTAL_DIST (DIST * NODES)

struct ant {
    int curNode, nextNode, pathIndex;
    int tabu[NODES];
    int solution[NODES];
    float solutionLen;
};

struct nodeTSP {
    float x, y;
};

using namespace std;
//内存数据
float* heuristic;
double* phero;
struct ant antColony[ANTS];
float bestSol[ANTS];
float globalBest = TOTAL_DIST;
hiprandState state[ANTS];
const size_t heuristic_size = sizeof(float) * size_t(NODES * NODES);
const size_t phero_size = sizeof(double) * size_t(NODES * NODES);
//GPU内存数据
float* heuristic_d;
double* phero_d;
struct ant* antColony_d;
float* bestSol_d;
hiprandState* state_d;
int BLOCKS, THREADS;

__global__ void initializeAnts(struct ant* antColony_d, hiprandState* state_d,
    float* bestSol_d);
__global__ void setuCurandStates(hiprandState* stated_d, unsigned long t,
    float* bestSol_d);
__global__ void restartAnts(struct ant* antColony_d, hiprandState* state_d,
    float* bestSol_d);
__global__ void constructSolution(struct ant* antColony_d, hiprandState* state_d, float* heuristic_d, double* phero_d);
__global__ void atomicUpdate(struct ant* antColony_d, double* phero_d);
__device__ double probFunctionProduct(int from, int to, double* phero_d, float* heuristic_d);
__device__ int NextNode(struct ant* antColony_d, int pos, float* heuristic_d, double* phero_d, hiprandState* state_d);

float euclideanDistance(float x1, float x2, float y1, float y2) {
    float xd = x1 - x2;
    float yd = y1 - y2;
    return (float)(sqrt(xd * xd + yd * yd));
}

void constructTSP(string graph, nodeTSP* nodes) {
    ifstream infile(("instances/" + graph + ".tsp").c_str());
    string line;
    bool euclidean = true;
    int node;
    float x, y;
    bool reading_nodes = false;

    while (getline(infile, line)) {
        istringstream iss(line);
        string word;
        if (!reading_nodes) {
            iss >> word;
            if (word.compare("EDGE_WEIGHT_TYPE") == 0) {
                iss >> word >> word;
                cout << "edge type: " << word << endl;
                euclidean = !word.compare("EUC_2D");
            }
            else if (word.compare("NODE_COORD_SECTION") == 0) {
                reading_nodes = true;
            }
        }
        else if (iss >> node >> x >> y) {
            nodes[node - 1].x = x;
            nodes[node - 1].y = y;
        }
    }
    infile.close();
    for (int from = 0; from < NODES; from++) {
        for (int to = from + 1; to < NODES; to++) {
            float edge_weight;
            if (euclidean) {
                edge_weight = euclideanDistance(nodes[from].x, nodes[to].x,
                    nodes[from].y, nodes[to].y);
            }

            if (edge_weight == 0) {
                edge_weight = 1.0;
            }
            heuristic[from + to * NODES] = edge_weight;
            heuristic[to + from * NODES] = edge_weight;
            phero[from + to * NODES] = PHERO_INITIAL;
            phero[to + from * NODES] = PHERO_INITIAL;
        }
    }
}

__global__ void setupCurandStates(hiprandState* state_d, unsigned long t) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(t, gid, 0, &state_d[gid]);
}

__global__ void initializeAnts(struct ant* antColony_d, hiprandState* state_d, float* bestSol_d) {

    int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
    for (int node = 0; node < NODES; node++) {

        antColony_d[ant_id].tabu[node] =
            0;
        antColony_d[ant_id].solution[node] =
            -1;
    }
    bestSol_d[ant_id] = (float)TOTAL_DIST;
    antColony_d[ant_id].curNode = hiprand(&state_d[ant_id]) % NODES;
    antColony_d[ant_id].solution[0] = antColony_d[ant_id].curNode;
    antColony_d[ant_id].tabu[antColony_d[ant_id].curNode] =
        1;
    antColony_d[ant_id].nextNode = -1;
    antColony_d[ant_id].solutionLen = 0;
    antColony_d[ant_id].pathIndex = 1;
}

__global__ void restartAnts(struct ant* antColony_d, hiprandState* state_d,
    float* bestSol_d) {

    int ant_id = blockDim.x * blockIdx.x + threadIdx.x;

    for (int node = 0; node < NODES; node++) {
        antColony_d[ant_id].tabu[node] =
            0;
        antColony_d[ant_id].solution[node] =
            -1;
    }
    if (antColony_d[ant_id].solutionLen < bestSol_d[ant_id] &&
        antColony_d[ant_id].solutionLen > 0) {
        bestSol_d[ant_id] = antColony_d[ant_id].solutionLen;

    }
    antColony_d[ant_id].curNode = hiprand(&state_d[ant_id]) % NODES;
    antColony_d[ant_id].solution[0] = antColony_d[ant_id].curNode;
    antColony_d[ant_id].tabu[antColony_d[ant_id].curNode] =
        1; 
    antColony_d[ant_id].nextNode = -1;
    antColony_d[ant_id].solutionLen = 0;
    antColony_d[ant_id].pathIndex = 1;
}

void acoSolve() {
    int iteration = 0;
    while (iteration++ < MAX_ITERATIONS) {
        constructSolution << <BLOCKS, THREADS >> > (antColony_d, state_d, heuristic_d,
            phero_d);

        hipDeviceSynchronize();
        // Move solution back to Host
        hipMemcpy(antColony, antColony_d, sizeof(antColony),
            hipMemcpyDeviceToHost);
        for (int from = 0; from < NODES; from++) {
            for (int to = 0; to < NODES; to++) {
                if (from != to) {
                    phero[from + to * NODES] *= (1.0 - RHO);
                    if (phero[from + to * NODES] < 0.0) {
                        phero[from + to * NODES] = PHERO_INITIAL;
                    }
                }
            }
        } 

        hipMemcpy(phero_d, phero, phero_size, hipMemcpyHostToDevice);
        hipMemcpy(bestSol, bestSol_d, sizeof(bestSol), hipMemcpyDeviceToHost);
        atomicUpdate << <BLOCKS, THREADS >> > (antColony_d, phero_d);

        for (int i = 0; i < ANTS; i++) {
            if (bestSol[i] < globalBest) {
                globalBest = bestSol[i];
            }
        }

        restartAnts << <BLOCKS, THREADS >> > (antColony_d, state_d, bestSol_d);
        hipDeviceSynchronize();

    }

    printf("Best Solution %f ", globalBest);
}

__global__ void atomicUpdate(struct ant* antColony_d, double* phero_d) {

    int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
    int from, to;
    for (int i = 0; i < NODES; i++) {
        from = antColony_d[ant_id].solution[i];
        if (i > NODES - 1) {
            to = antColony_d[ant_id].solution[i + 1];
        }
        else {
            to = antColony_d[ant_id].solution[0];
        }
        atomicAdd(&phero_d[from + to * NODES], Q / antColony_d[ant_id].solutionLen * RHO);
        atomicAdd(&phero_d[from + to * NODES], Q / antColony_d[ant_id].solutionLen * RHO);
    }
}
__global__ void constructSolution(struct ant* antColony_d, hiprandState* state_d,
    float* heuristic_d, double* phero_d) {

    int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
    int node = 0;

    while (node++ < NODES) {
        if (antColony_d[ant_id].pathIndex < NODES) {
            antColony_d[ant_id].nextNode =
                NextNode(antColony_d, ant_id, heuristic_d, phero_d, state_d);
            antColony_d[ant_id].tabu[antColony_d[ant_id].nextNode] = 1;
            antColony_d[ant_id].solution[antColony_d[ant_id].pathIndex++] =
                antColony_d[ant_id].nextNode;
            antColony_d[ant_id].solutionLen +=
                heuristic_d[antColony_d[ant_id].curNode +
                (antColony_d[ant_id].nextNode * NODES)];

            if (antColony_d[ant_id].pathIndex == NODES) {
                antColony_d[ant_id].solutionLen +=
                    heuristic_d[antColony_d[ant_id].solution[NODES - 1] +
                    (antColony_d[ant_id].solution[0] * NODES)];
            }
            antColony_d[ant_id].curNode = antColony_d[ant_id].nextNode;
        }
    }
}

__device__ double probFunctionProduct(int from, int to, double* phero_d,
    float* heuristic_d) {
    double result;
    result = pow(phero_d[from + to * NODES], ALPHA) *
        pow(1 / (heuristic_d[from + to * NODES]), BETA);
    if (!isnan(result)) {
        return (double)((result));
    }
    else {
        return 0;
    }
}

__device__ int NextNode(struct ant* antColony_d, int pos, float* heuristic_d,
    double* phero_d, hiprandState* state_d) {
    int to, from;
    double denom = 0.00000001;
    from = antColony_d[pos].curNode;
    for (to = 0; to < NODES; to++) {
        if (antColony_d[pos].tabu[to] == 0) {
            denom += probFunctionProduct(from, to, phero_d, heuristic_d);
        }
    }
    assert(denom != 0.0);
    to++;
    int count = NODES - antColony_d[pos].pathIndex;
    do {
        double p;
        to++;
        if (to >= NODES)
            to = 0;
        if (antColony_d[pos].tabu[to] ==
            0) {
            p = probFunctionProduct(from, to, phero_d, heuristic_d) / denom;
            double x = (double)(hiprand(&state_d[pos]) % 1000000000) / 1000000000.0;
            if (x < p) {
                break;
            }
            count--;
            if (count == 0) {
                break;
            }
        }
    } while (1);
    return to;
}

int main() {
    float exec_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (ANTS <= 1024) {
        BLOCKS = 1;
        THREADS = ANTS;
    }
    else {
        THREADS = 1024;
        BLOCKS = ceil(ANTS / (float)THREADS);
    }
    heuristic = (float*)malloc(NODES * NODES * sizeof(float));
    phero = (double*)malloc(NODES * NODES * sizeof(double));

    nodeTSP nodes[NODES];
    constructTSP("lin105", nodes);
    long long time0, time1, freq;
    QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
    QueryPerformanceCounter((LARGE_INTEGER*)&time0);
    hipMalloc((void**)&antColony_d, sizeof(antColony));
    hipMalloc((void**)&state_d, sizeof(state));
    hipMalloc((void**)&bestSol_d, sizeof(bestSol));
    hipMalloc((void**)&heuristic_d, heuristic_size);
    hipMalloc((void**)&phero_d, phero_size);

    hipMemcpy(heuristic_d, heuristic, heuristic_size, hipMemcpyHostToDevice);
    hipMemcpy(phero_d, phero, phero_size, hipMemcpyHostToDevice);

    time_t t;
    time(&t);
    setupCurandStates << <BLOCKS, THREADS >> > (state_d, (unsigned long)t);
    hipDeviceSynchronize();
    initializeAnts << <BLOCKS, THREADS >> > (antColony_d, state_d, bestSol_d);
    hipDeviceSynchronize();

    hipEventRecord(start, 0);
    acoSolve();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&exec_time, start, stop);

    printf("%5.5f \n", exec_time / 1000);
    free(phero);
    free(heuristic);

    hipFree(antColony_d);
    hipFree(heuristic_d);
    hipFree(phero_d);
    hipFree(state_d);
    hipFree(bestSol_d);
    QueryPerformanceCounter((LARGE_INTEGER*)&time1);
    cout << "GPU加速算法时间为" << (time1 - time0) * 1000.0 / freq << "ms" << endl;
    return 0;
}
